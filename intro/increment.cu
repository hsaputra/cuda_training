// simple increment kernel

#include <hip/hip_runtime.h>
#include <stdio.h>

//TODO: increment kernel
__global__
void increment(float *val) {
  *val += 2.0f; 
}

int main(void)
{
    // create host array and initialize
    float *device_pointer;
 
    // print original value
    float input = 40.0f;    
    printf("Input: %f\n", input); 

    // allocate device memory
    hipMalloc(&device_pointer, sizeof(float));

    // memcpy to device
    hipMemcpy(device_pointer, &input, sizeof(float), hipMemcpyHostToDevice);

    // launch the increment kernel
    increment<<<1, 1>>>(device_pointer);

    // memcpy results back to host
    hipMemcpy(&input, device_pointer, sizeof(float), hipMemcpyDeviceToHost);

    // print new value
    printf("New Input: %f\n", input);

    return 0;
}
