
#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>

using std::vector;

// parameter describing the size of matrix A
const int rows = 4096;
const int cols = 4096;

const int BLOCK_SIZE = 32;

// naive transpose kernel
__global__ void matrixTransposeNaive(float *_a,   // pointer to matrix A on the device
                                     float *_b)   // pointer to matrix B on the device
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // row
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // col

    int index_in = i*cols+j;   // (i,j) from matrix A
    int index_out = j*rows+i;  // becomes (j,i) in matrix B = transpose(A)

    _b[index_out] = _a[index_in];
}

// coalesced memory transpose kernel
__global__ void matrixTransposeShared(float *_a,   // pointer to matrix A on the device
                                      float *_b)   // pointer to matrix B on the device
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index_in = j*cols+i;   // (i,j) from matrix A

    // this thread fills in the appropriate box inside the shared memory in this block
    __shared__ float tile[BLOCK_SIZE][BLOCK_SIZE];
    tile [ threadIdx.x ] [ threadIdx.y ] = _a [index_in];

    // wait until all threads in this block are done writing to shared memory in parallel
    __syncthreads();

    i = blockIdx.y * blockDim.x + threadIdx.x;
    j = blockIdx.x * blockDim.y + threadIdx.y;
    int index_out = j*rows+i;  // (i,j) from matrix A becomes (j,i) in matrix B = transpose(A)

    _b[index_out] = tile[ threadIdx.y ] [ threadIdx.x ]; 
}

// coalesced memory transpose kernel without banking conflicts
__global__ void matrixTransposeNoBankConflicts(float *_a,   // pointer to matrix A on the device
                                      float *_b)   // pointer to matrix B on the device
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index_in = j*cols+i;   // (i,j) from matrix A

    // this thread fills in the appropriate box inside the shared memory in this block
    __shared__ float tile[BLOCK_SIZE][BLOCK_SIZE+1];
    tile [ threadIdx.x ] [ threadIdx.y ] = _a [index_in];

    i = blockIdx.y * blockDim.x + threadIdx.x;
    j = blockIdx.x * blockDim.y + threadIdx.y;
    int index_out = j*rows+i;  // (i,j) from matrix A becomes (j,i) in matrix B = transpose(A)

    // wait until all threads in this block are done writing to shared memory in parallel
    __syncthreads();

    _b[index_out] = tile[ threadIdx.y ] [ threadIdx.x ]; 
}

// the main program starts life on the CPU and calls device kernels as required
int main(int argc, char *argv[])
{
    // allocate space in the host for storing input arrays (a and b) and the output array (c)
    vector<float> a(rows*cols);
    vector<float> b(rows*cols);

    // define device pointers for the same arrays when they'll be copied to the device
    float *_a, *_b;

    // allocate memory on the device (GPU) and check for errors (if any) during this call
    hipError_t err;

    // allocate space for matrix A 
    if (err = hipMalloc((void **) &_a, rows*cols*sizeof(float))) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    // allocate space for matrix B
    if (err = hipMalloc((void **) &_b, rows*cols*sizeof(float))) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Fill matrix A
    for (int row = 0; row < rows; row++) {
        for (int col = 0; col < cols; col++) {
            a[row + col*rows] = row + col*rows;
        }
    }

    // Copy array contents of A from the host (CPU) to the device (GPU)
    // Note that this is copied to the "global" memory on the device and is accessible to all threads in all blocks
    hipMemcpy(_a, a.data(), rows*cols*sizeof(float), hipMemcpyHostToDevice);

    // assign a 2D distribution of 16 x 16 x 1 CUDA "threads" within each CUDA "block"
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    
    // calculate number of blocks along X and Y in a 2D CUDA "grid"
    dim3 dimGrid( ceil(float(rows)/float(dimBlock.x)), ceil(float(cols)/float(dimBlock.y)), 1 );

    float time;

    // create CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start the timer
    hipEventRecord( start, 0);

    // launch the GPU kernel

    // cudaMemcpy(_b, _a, cols*rows*sizeof(float), cudaMemcpyDeviceToDevice);
    // matrixTransposeNaive<<<dimGrid,dimBlock>>>(_a, _b);
    // matrixTransposeShared<<<dimGrid,dimBlock>>>(_a, _b);
    matrixTransposeNoBankConflicts<<<dimGrid,dimBlock>>>(_a, _b);

    // stop the timer
    hipEventRecord( stop, 0);
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time, start, stop);

    // print out the time required for the kernel to finish the transpose operation
    double data = 2.0 * (rows * cols * sizeof(float)) / (1024 * 1024 * 1024); 
    printf("data %f \n", data);
    printf("time %f \n", time/1000);
    double Bandwidth = data/(time/1000);
    printf("Elapsed Time  = %f ms Bandwidth achieved (GB/s) = %f\n", time, Bandwidth);

    // copy the answer back to the host (CPU) from the device (GPU)
    hipMemcpy(b.data(), _b, cols*rows*sizeof(float), hipMemcpyDeviceToHost);

    // for(int i = 0; i < 64; i++) {
    //   for(int j = 0; j < 64; j++) {
    //     printf("%f ", b[i * rows + j]);
    //   }
    //   printf("\n");
    // }

    // free device memory
    hipFree(_a);
    hipFree(_b);

    // successful program termination
    return 0;
}
