#include "hip/hip_runtime.h"
// y=convolve(x,k)

#include <stdio.h>
#include <stdlib.h>  // rand(), RAND_MAX

#include "../common.h"


__global__
static void kernel(float *d_y, int nx, const float *d_x, int nk, const float *d_k)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    if (x >= nx) return;

    float sum = 0;

    // TODO convolution (dot product)
    //   y[0] = x[0]*k[0] + x[1]*k[1] + x[2]*k[2]
    //   y[1] = x[1]*k[0] + x[2]*k[1] + x[3]*k[2]
    //   y[2] = x[2]*k[0] + x[3]*k[1] + x[4]*k[2]
    //   y[3] = x[3]*k[0] + x[4]*k[1] + x[5]*k[2]
    //   ...

    // several extensions to play:
    // TODO textured cached read of d_x (tex1Dfetch)
    // TODO shared mem cached read of d_x
    // TODO __constant__ read of d_k   (hipMemcpyToSymbol)

    // TODO template<nk>

    d_y[x] = sum;
}

int main()
{
    int nx = 5;
    int nk = 3;

    float *h_x = (float *)malloc(nx*sizeof(*h_x));
    for (int i = 0; i < nx; ++i)
        h_x[i] = 1; //10 * float(rand())/RAND_MAX; // [0-10]
    float *h_k = (float *)malloc(nk*sizeof(*h_k));
    for (int i = 0; i < nk; ++i)
        h_k[i] = 1; // constant kernel: 1

    // print inputs
    printf("x:    ");
    for (int i = 0; i < min(13,nx); ++i)
        printf("%6.2f ", h_x[i]);
    printf("\nk:    ");
    for (int i = 0; i < min(13,nk); ++i)
        printf("%6.2f ", h_k[i]);
    putchar('\n');

    // allocate and populate input (all ones)
    float *d_x, *d_k, *d_y;
    size_t xbytes = nx*sizeof(*d_x);
    size_t kbytes = nk*sizeof(*d_k);
    CUDA(hipMalloc(&d_x, xbytes));
    CUDA(hipMalloc(&d_k, kbytes));
    CUDA(hipMalloc(&d_y, xbytes));
    CUDA(hipMemcpy(d_x,  h_x, xbytes, hipMemcpyHostToDevice));
    CUDA(hipMemcpy(d_k,  h_k, kbytes, hipMemcpyHostToDevice));
    CUDA(hipMemset(d_y,  0,   xbytes)); // zero-out output

    // create events
    hipEvent_t start, stop;
    CUDA(hipEventCreate(&start));
    CUDA(hipEventCreate(&stop));

    // filter
    CUDA(hipEventRecord(start, 0));
    int threads = 256;
    kernel<<<divup(nx,threads), threads>>>(d_y, nx, d_x, nk, d_k);
    CUDA(hipGetLastError());
    CUDA(hipEventRecord(stop, 0));

    // time kernel
    CUDA(hipEventSynchronize(stop));
    float time_ms = 0;
    CUDA(hipEventElapsedTime(&time_ms, start, stop));

    // print upper-left corner to verify
    size_t bytes = nx*sizeof(*d_y); // max needed since only care about corner
    float *h_y = (float *)malloc(bytes);
    CUDA(hipMemcpy(h_y, d_y, bytes, hipMemcpyDeviceToHost));
    printf("y:    ");
    for (int i = 0; i < nx; ++i) {
        if (i < 13)  printf("%6.2f ", h_y[i]);
        if ((i < nx - nk && h_y[i] != nk) || (nx - nk <= i && h_y[i] != nx - i)) {
            printf("\nFAIL at i=%d!", i);
            break;
        }
    }
    putchar('\n');

    double flops = nx * 2*nk; // (nk) multiply and add ops for each input (nx)
    printf("performance: %f GFLOP/s\n", (flops/1e9) / (time_ms/1e3));

    return 0;
}
