#include "hip/hip_runtime.h"
/*

Solution of the Laplace equation for heat conduction in a square plate

*/

#include <iostream>

// global variables

const int NX = 4096;      // mesh size (number of node points along X)
const int NY = 4096;      // mesh size (number of node points along Y)
    
const int MAX_ITER=1000;  // number of Jacobi iterations

// device function to update the array T_new based on the values in array T_old
// note that all locations are updated simultaneously on the GPU 
__global__ void Laplace(double *T_old, double *T_new)
{
    // TODO compute the "i" and "j" location of the node point handled by this thread

    // TODO get the natural index values of node (i,j) and its neighboring nodes

    // TODO update "interior" node points
}

// initialization

void Initialize(double *TEMPERATURE)
{
    for(int i=0;i<NX;i++) {
        for(int j=0;j<NY;j++) {
            int index = i + j*NX;
            TEMPERATURE[index]=0.0;
        }
    }

    // set left wall to 1

    for(int j=0;j<NY;j++) {
        int index = j*NX;
        TEMPERATURE[index]=1.0;
    }
}

int main(int argc,char **argv)
{
    double *_T1, *_T2;  // pointers to device (GPU) memory

    // allocate a "pre-computation" T array on the host
    double *T = new double [NX*NY];

    // initialize array on the host
    Initialize(T);

    // allocate storage space on the GPU
    hipMalloc((void **)&_T1,NX*NY*sizeof(double));
    hipMalloc((void **)&_T2,NX*NY*sizeof(double));

    // copy (initialized) host arrays to the GPU memory from CPU memory
    hipMemcpy(_T1,T,NX*NY*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(_T2,T,NX*NY*sizeof(double),hipMemcpyHostToDevice);

    // assign a 2D distribution of CUDA "threads" within each CUDA "block"    
    int ThreadsPerBlock=16;
    dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );

    // calculate number of blocks along X and Y in a 2D CUDA "grid"
    dim3 dimGrid( ceil(double(NX)/double(dimBlock.x)), ceil(double(NY)/double(dimBlock.y)), 1 );

    // begin Jacobi iteration
    int k = 0;
    while(k<MAX_ITER) {
        Laplace<<<dimGrid, dimBlock>>>(_T1,_T2);   // update T1 using data stored in T2
        Laplace<<<dimGrid, dimBlock>>>(_T2,_T1);   // update T2 using data stored in T1
        k+=2;
    }

    // copy final array to the CPU from the GPU 
    hipMemcpy(T,_T2,NX*NY*sizeof(double),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
/*
    // print the results to screen
    for (int j=NY-1;j>=0;j--) {
        for (int i=0;i<NX;i++) {
            int index = i + j*NX;
            std::cout << T[index] << " ";
        }
        std::cout << std::endl;
    }
*/
    // release memory on the host 
    delete T;

    // release memory on the device 
    hipFree(_T1);
    hipFree(_T2);

    return 0;
}
